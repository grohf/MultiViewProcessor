#include "hip/hip_runtime.h"
/*
 * RigidBodyTransformationAdvancedEstimatior.cpp
 *
 *  Created on: Nov 20, 2012
 *      Author: avo
 */

#include "RigidBodyTransformationAdvancedEstimatior.h"

#include <hiprand.h>

#include <hip/hip_runtime_api.h>
#include <helper_image.h>

#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/random.h>

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "point_info.hpp"


#include "../sink/pcd_io.h"
#include "utils.hpp"
#include "device_utils.hpp"


namespace device
{

	struct RigidBodyTransformationBaseKernel : public FeatureBaseKernel
	{
		enum
		{
			corresp_list_length = 8,
		};
	};


	struct CorrespondenceListEstimator : public RigidBodyTransformationBaseKernel
	{
		enum
		{
			local_points_per_block = 32, //local_points_per_block
			global_points_per_sweep = 32, //global_points_per_sweep
			dx = local_points_per_block*global_points_per_sweep,
			minDelta = 10000,
		};


		float *input_feature_histo;
		int *input_idxList;
		unsigned int *input_idxLength;
		float *input_rndList;

		unsigned int *output_s_idxList;

		float *output_corresp_prob;
		unsigned int *output_corresp_idx;

		unsigned int n_view;

		__device__ __forceinline__ void compareDiv(
			float  &val1,
			float  &val2,
			unsigned int  &idx1,
			unsigned int  &idx2,
			unsigned int  dir
		) const
		{
		    float tmp_val;
		    unsigned int tmp_idx;

		    if ((val1 > val2) == dir)
		    {
		    	tmp_val = val1;
		        val1 = val2;
		        val2 = tmp_val;
		        tmp_idx = idx1;
		        idx1 = idx2;
		        idx2 = tmp_idx;
		    }
		}

		__device__ __forceinline__ void
		operator () () const
		{

			__shared__ float shm_local_histo[local_points_per_block*bins];
			__shared__ unsigned int shm_local_idx[local_points_per_block];

			__shared__ float shm_global_histo[global_points_per_sweep*bins];
			__shared__ unsigned int shm_global_idx[global_points_per_sweep];

			//TODO: extra buffer for list
			__shared__ float shm_dist_buffer[local_points_per_block*global_points_per_sweep*2];
			__shared__ unsigned int shm_idx_buffer[local_points_per_block*global_points_per_sweep*2];

//			__shared__ float shm_dist_sorted[local_points_per_block*corresp_list_length];
//			__shared__ unsigned int shm_idx_sorted[local_points_per_block*corresp_list_length];

			__shared__ unsigned int view_src;
			__shared__ unsigned int view_target;

			unsigned int tid = threadIdx.x;

			if(tid==0)
			{
				unsigned int x = blockIdx.z;
				unsigned int y = 0;
				unsigned int i = 1;
				while(x >= n_view - i)
				{
					x -= n_view-i;
					i++;
					y++;
				}
				view_src = y;
				view_target = x+y+1;
			}
			__syncthreads();

//			if(blockIdx.x==0 && tid==0)
//			{
//				printf("(%d) - > src: %d target: %d \n",blockIdx.z,view_src,view_target);
//			}

			if(threadIdx.x<local_points_per_block)
			{
				float f = input_rndList[(blockIdx.z*gridDim.x+blockIdx.x)*local_points_per_block+threadIdx.x];
				unsigned int begin = (view_src>0)?input_idxLength[view_src-1]:0;
				unsigned int length = input_idxLength[view_src] - begin;
				unsigned int idx = input_idxList[begin+(unsigned int)(f*length)];

//				if(tid==0)
//					printf("begin: %d | length: %d \n",begin,length);

				idx -= (view_src*640*480);
//				unsigned int oy = idx/640;
//				unsigned int ox = idx - oy*640;
//				printf("(%d) %f -> %d -> %d / %d \n",tid,f,idx,oy,ox);

				shm_local_idx[threadIdx.x] = idx;
				output_s_idxList[(blockIdx.z*gridDim.x+blockIdx.x)*local_points_per_block+threadIdx.x] = idx;

			}
			__syncthreads();

			for(int i=threadIdx.x; i<local_points_per_block*bins; i+=blockDim.x)
			{
				unsigned int b = i/local_points_per_block;
				unsigned int sl = i-b*local_points_per_block;
				shm_local_histo[b*local_points_per_block + sl] = input_feature_histo[view_src*640*480*bins_n_meta+b*640*480+shm_local_idx[sl]];
			}

			//TODO: delete
			__syncthreads();
/*			if(blockIdx.x==0 && tid==7)
			{
				for(int p=0;p<local_points_per_block;p++)
				{
					float sum = 0.f;
					printf("L: (%d -> %d): ",p,shm_local_idx[p]);
					for(int f=0;f<features;f++)
					{
						sum = 0.f;
						for(int b=0;b<bins_per_feature;b++)
						{
							float tmp = shm_local_histo[(f*bins_per_feature+b)*local_points_per_block+p];
							printf("%f ",tmp);
							sum += tmp;
						}
						printf(" (sum: %f) || ",sum);
					}
					printf("\n");
				}
			}
*/

			unsigned int target_begin = input_idxLength[view_target-1];
			unsigned int target_length = input_idxLength[view_target]-target_begin;
			if(blockIdx.x==0 && tid==0)
				printf("target length: %d \n",target_length);
			//TODO: handle rest points
			for(int l=0;l<target_length/global_points_per_sweep;l++)
//			for(int l=0;l<2;l++)
			{

				//Get the index of the k global points;
				for(int i=threadIdx.x;i<global_points_per_sweep;i+=blockDim.x)
				{
					shm_global_idx[i] = input_idxList[target_begin+l*global_points_per_sweep+i]-view_target*640*480;
				}
				__syncthreads();

				for(int i=threadIdx.x; i<global_points_per_sweep*bins; i+=blockDim.x)
				{
					unsigned int b = i/global_points_per_sweep;
					unsigned int sl = i - b*global_points_per_sweep;

					shm_global_histo[b*global_points_per_sweep+sl] = input_feature_histo[view_target*640*480*bins_n_meta+b*640*480+shm_global_idx[sl]];
				}
				__syncthreads();

				for(int i=threadIdx.x; i<global_points_per_sweep*local_points_per_block; i+=blockDim.x)
				{
					unsigned int wl = i/local_points_per_block;
					shm_idx_buffer[local_points_per_block*global_points_per_sweep+i] = shm_global_idx[wl];
				}
				__syncthreads();

/*
				__syncthreads();
				if(blockIdx.x==0 && tid==7)
				{
					for(int p=0;p<global_points_per_sweep;p++)
					{
						float sum = 0.f;
						printf("(G: %d -> %d): ",p,shm_global_idx[p]);
						for(int f=0;f<features;f++)
						{
							sum = 0.f;
							for(int b=0;b<bins_per_feature;b++)
							{
								float tmp = shm_global_histo[(f*bins_per_feature+b)*global_points_per_sweep+p];
								printf("%f ",tmp);
								sum += tmp;
							}
							printf(" (sum: %f) || ",sum);
						}
						printf("\n");
					}
				}
*/

				unsigned int lid = tid/local_points_per_block;
				unsigned int ltid = tid - lid*local_points_per_block;
				float div = 0.f;
//				div = klEuclideanDivergence(&(shm_local_histo[ltid]),&(shm_global_histo[lid]),features,bins_per_feature,local_points_per_block,global_points_per_sweep);
//				div = chiSquaredDivergence(&(shm_local_histo[ltid]),&(shm_global_histo[lid]),features,bins_per_feature,local_points_per_block,global_points_per_sweep);
				div = chiSquaredEuclideanDivergence(&(shm_local_histo[ltid]),&(shm_global_histo[lid]),features,bins_per_feature,local_points_per_block,global_points_per_sweep);
				shm_dist_buffer[local_points_per_block*global_points_per_sweep+lid*local_points_per_block+ltid] = div;

//				__syncthreads();


				//Bitonic sort Buffer
				for(int size = 2; size < global_points_per_sweep; size <<= 1)
				{
					unsigned int dir = 1 ^ ((lid & (size/2)) != 0);
					for(int stride = size/2; stride > 0; stride >>= 1)
					{
						__syncthreads();
						if(lid<global_points_per_sweep/2)
						{
							unsigned int pos = (2 * lid - (lid & (stride - 1))) * 32 + ltid + local_points_per_block*global_points_per_sweep;
							compareDiv(shm_dist_buffer[pos], shm_dist_buffer[pos+stride*32], shm_idx_buffer[pos], shm_idx_buffer[pos+stride*32], dir);
						}
					}
				}

				for(int stride = global_points_per_sweep/2; stride > 0; stride >>= 1)
				{
					__syncthreads();
					if(lid<global_points_per_sweep/2)
					{
						unsigned int pos = (2 * lid - (lid & (stride - 1))) * 32 + ltid + local_points_per_block*global_points_per_sweep;
						compareDiv(shm_dist_buffer[pos], shm_dist_buffer[pos+stride*32], shm_idx_buffer[pos], shm_idx_buffer[pos+stride*32], 1);
					}
				}
//				__syncthreads();
//				if(blockIdx.x==0 && tid==7)
//				{
//					for(int j=0;j<local_points_per_block;j++)
//					{
//						bool fine = true;
//						float old = 0.f;
//						printf("(%d): ",j);
//						for(int g=0;g<global_points_per_sweep;g++)
//						{
//							fine = (old <= shm_dist_buffer[g*local_points_per_block+j]);
//							printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//							old = shm_dist_buffer[g*local_points_per_block+j];
//							if(!fine)
//								printf("! (%d/%d) ! ",j,g);
//						}
//						printf(" \n");
//						fine = true;
//					}
//				}

				__syncthreads();
				if(l==0)
				{
					for(int i=threadIdx.x; i<local_points_per_block*global_points_per_sweep; i+=blockDim.x)
					{
						shm_dist_buffer[i] = shm_dist_buffer[local_points_per_block*global_points_per_sweep+i];
						shm_idx_buffer[i] = shm_idx_buffer[local_points_per_block*global_points_per_sweep+i];
					}
				}
				else
				{
					// Merge 2 sorted lists together
					unsigned int stride = global_points_per_sweep;
					unsigned int offset = lid & (stride - 1);
//
					__syncthreads();

					unsigned int pos = (2 * lid - (lid & (stride - 1)))*32 + ltid;
					compareDiv(shm_dist_buffer[pos], shm_dist_buffer[pos+stride*32], shm_idx_buffer[pos], shm_idx_buffer[pos+stride*32],1);

					for(stride >>= 1; stride > 0; stride >>=1)
					{
						__syncthreads();
						unsigned int pos = (2 * lid - (lid & (stride - 1)))*32 + ltid;
						if(offset >= stride)
						{
							compareDiv(shm_dist_buffer[pos-stride*32], shm_dist_buffer[pos], shm_idx_buffer[pos-stride*32], shm_idx_buffer[pos],1);
						}

					}
				}

//				__syncthreads();
//				if(blockIdx.x==0 && tid==7)
//				{
//					if(l%100==0)
//						for(int j=0;j<1;j++)
//						{
//							bool fine = true;
//							float old = 0.f;
//							printf("(%d/%d): ",l,j);
//							for(int g=0;g<global_points_per_sweep;g++)
//							{
//								fine = (old <= shm_dist_buffer[g*local_points_per_block+j]);
//								printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//								old = shm_dist_buffer[g*local_points_per_block+j];
//								if(!fine)
//									printf("! (%d/%d) ! ",j,g);
//							}
//							printf(" \n");
//							for(int g=0;g<global_points_per_sweep;g++)
//							{
//								printf(" %d ",shm_idx_buffer[g*local_points_per_block+j]);
//							}
//							printf(" \n");
//							fine = true;
//						}
//				}

			}
			__syncthreads();


//			__syncthreads();
//			if(blockIdx.x==0 && tid==7)
//			{
//					for(int j=0;j<8;j++)
//					{
//						printf("(%d): ",j);
//						for(int g=0;g<global_points_per_sweep;g++)
//						{
//							printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//
//						}
//						printf(" \n");
//						for(int g=0;g<global_points_per_sweep;g++)
//						{
//							printf(" %d ",shm_idx_buffer[g*local_points_per_block+j]);
//						}
//						printf(" \n");
//					}
//			}

//			if(blockIdx.x==0 && tid==7)
//			{
//					for(int j=0;j<1;j++)
//					{
//						printf("(error: %d): ",j);
//						for(int g=0;g<corresp_list_length;g++)
//						{
//							printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//
//						}
//						printf("\n");
//					}
//					printf("---------------------------- \n");
//			}

//			for(int i=threadIdx.x; i<local_points_per_block*corresp_list_length; i+=blockDim.x)
//			{
//				unsigned int l = i/local_points_per_block;
//				unsigned int p = i-l*local_points_per_block;
//
//				float errorDelta = shm_dist_buffer[p*local_points_per_block+l];
//
//				if(errorDelta < 1.f/minDelta)
//					errorDelta = 1.f/minDelta;
//
//				shm_dist_buffer[p*local_points_per_block+l] = 1.f/(errorDelta);
//			}
//
//			__syncthreads();
//
//			if(blockIdx.x==0 && tid==7)
//			{
//					for(int j=0;j<1;j++)
//					{
//						printf("(prob: %d): ",j);
//						for(int g=0;g<corresp_list_length;g++)
//						{
//							printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//
//						}
//						printf("\n");
//					}
//					printf("---------------------------- \n");
//			}

			if(tid<local_points_per_block)
			{
				float cumSum = 0.f;
				for(int i=0;i<corresp_list_length;i++)
				{
					float tmp = shm_dist_buffer[i*local_points_per_block+tid];
					tmp = (tmp>0)?1.f/(tmp*tmp):1.f/minDelta;
					shm_dist_buffer[i*local_points_per_block+tid] = cumSum += tmp;
				}
			}

			__syncthreads();

//			if(blockIdx.x==0 && tid==7)
//			{
//					for(int j=0;j<1;j++)
//					{
//						printf("(cum: %d): ",j);
//						for(int g=0;g<corresp_list_length;g++)
//						{
//							printf(" %f ",shm_dist_buffer[g*local_points_per_block+j]);
//
//						}
//						printf("\n");
//					}
//					printf("---------------------------- \n");
//			}


			for(int i=threadIdx.x; i<local_points_per_block*corresp_list_length; i+=blockDim.x)
			{
				unsigned int l = i/local_points_per_block;
				unsigned int p = i-l*local_points_per_block;

				output_corresp_prob[(blockIdx.z*gridDim.x+blockIdx.x+p)*local_points_per_block + l] = shm_dist_buffer[p*local_points_per_block+l]/shm_dist_buffer[(corresp_list_length-1)*local_points_per_block+l];
				output_corresp_idx[(blockIdx.z*gridDim.x+blockIdx.x+p)*local_points_per_block + l] = shm_idx_buffer[p*local_points_per_block+l];
			}

		}
	};
	__global__ void estimateBestCorrespondences(const CorrespondenceListEstimator cle){ cle(); }


	struct TransformationBaseKernel : public RigidBodyTransformationBaseKernel
	{
		enum
		{
			n_corresp = 4,
			n_combinations = (n_corresp*(n_corresp-1))/2,

			groups_per_warp = WARP_SIZE/n_corresp,
			group_length = WARP_SIZE/groups_per_warp,
		};
	};


	struct CombinationErrorListEstimator : public TransformationBaseKernel
	{
		enum
		{
			dx = 1024,

			n_transformations = dx/n_corresp,
		};


		float4 			*input_pos;
		unsigned int 	*input_src_idx_corresp;
		unsigned int 	*input_target_idx_corresp;
		float 			*input_target_prob_corresp;

		float 			*input_src_rnd;
		float			*input_target_rnd;

		float			*output_combinationError;
		unsigned int	*output_combinationIdx_src;
		unsigned int	*output_combinationIdx_target;

		unsigned int s_length;
		unsigned int n_view;
		unsigned int combinationListOffset;

		__device__ __forceinline__ void
		operator () () const
		{


			__shared__ float combLength[n_transformations*n_combinations];
			__shared__ float points[dx*6];

			__shared__ unsigned int view_src;
			__shared__ unsigned int view_target;

			unsigned int tid = threadIdx.x;

			if(tid==0)
			{
				unsigned int x = blockIdx.z;
				unsigned int y = 0;
				unsigned int i = 1;
				while(x >= n_view - i)
				{
					x -= n_view-i;
					i++;
					y++;
				}
				view_src = y;
				view_target = x+y+1;
			}
			__syncthreads();


			unsigned int idx = (unsigned int)(input_src_rnd[(blockIdx.z*gridDim.x+blockIdx.x)*dx+tid]*s_length);
			unsigned int pidx = input_src_idx_corresp[idx];
			output_combinationIdx_src[(blockIdx.z*gridDim.x+blockIdx.x)*dx+tid] = pidx;
			float4 tmp = input_pos[view_src*640*480+pidx];

			points[dx*0 + tid] = tmp.x;
			points[dx*1 + tid] = tmp.y;
			points[dx*2 + tid] = tmp.z;


			float rnd_t = input_target_rnd[(blockIdx.z*gridDim.x+blockIdx.x)*dx+tid];
			unsigned int g = 0;
			while( rnd_t > input_target_prob_corresp[view_target*s_length*corresp_list_length + g*corresp_list_length + idx] )
			{
				g++;
			}

			pidx = input_target_idx_corresp[view_target*s_length*corresp_list_length + g*corresp_list_length + idx];
			output_combinationIdx_target[(blockIdx.z*gridDim.x+blockIdx.x)*dx+tid] = pidx;

			tmp = input_pos[view_target*640*480+pidx];
			points[dx*3 + tid] = tmp.x;
			points[dx*4 + tid] = tmp.y;
			points[dx*5 + tid] = tmp.z;

			__syncthreads();


			unsigned int wid = tid/WARP_SIZE;
			unsigned int wtid = tid - wid * WARP_SIZE;
			unsigned int gid = wtid/group_length;
			unsigned int gtid = wtid - gid * group_length;

			for(int i=n_corresp-1;i>0;i++)
			{
				if(gtid<i)
				{

					//to be continued...
				}
			}

		}
	};

	struct TransformationMatrixEstimator : public TransformationBaseKernel
	{
		enum
		{
			threads = 512,
//			n_corresp = 4,
//			n_combinations = (n_corresp*(n_corresp-1))/2,
			n_matrices = threads/n_corresp,

			groups_per_warp = WARP_SIZE/n_corresp,
			group_length = WARP_SIZE/groups_per_warp,
		};

		float4 			*input_pos;
		unsigned int 	*input_src_idx_corresp;
		unsigned int 	*input_target_idx_corresp;
		float 			*input_target_prob_corresp;

		float 			*input_src_rnd;
		float			*input_target_rnd;

		float			*output_transformationMatrices;

		unsigned int s_length;
		unsigned int n_view;

		__device__ __forceinline__
		unsigned int hash(unsigned int a) const
		{
			a = (a+0x7ed55d16) + (a<<12);
			a = (a^0xc761c23c) ^ (a>>19);
			a = (a+0x165667b1) + (a<<5);
			a = (a+0xd3a2646c) ^ (a<<9);
			a = (a+0xfd7046c5) + (a<<3);
			a = (a^0xb55a4f09) ^ (a>>16);
			return a;
		}


		__device__ __forceinline__ void
		operator () () const
		{
			__shared__ float points[threads*6*2];
			__shared__ float combLength[n_matrices*n_combinations];
			__shared__ float buffer[threads];

			__shared__ float centroids[n_matrices*6];
			__shared__ float matricesH[n_matrices*9];

			__shared__ unsigned int view_src;
			__shared__ unsigned int view_target;

			unsigned int tid = threadIdx.x;

			if(tid==0)
			{
				unsigned int x = blockIdx.z;
				unsigned int y = 0;
				unsigned int i = 1;
				while(x >= n_view - i)
				{
					x -= n_view-i;
					i++;
					y++;
				}
				view_src = y;
				view_target = x+y+1;
			}
			__syncthreads();


			unsigned int seed = hash(tid);
			 // seed a random number generator
			thrust::default_random_engine rng(seed);

			// create a mapping from random numbers to [0,1)
			thrust::uniform_real_distribution<float> u01(0,1);

			float x = u01(rng);
			float y = u01(rng);

			if(blockIdx.x==0 && tid==0)
				printf("%f %f \n",x,y);




/*
			unsigned int idx = (unsigned int)(input_src_rnd[blockIdx.x*threads+tid]*s_length);
			unsigned int pidx = input_src_idx_corresp[idx];
			float4 tmp = input_pos[view_src*640*480+pidx];

			points[threads*0 + tid] = tmp.x;
			points[threads*1 + tid] = tmp.y;
			points[threads*2 + tid] = tmp.z;

			pidx = input_target_idx_corresp[view_target*s_length*corresp_list_length + (unsigned int)(input_target_rnd[blockIdx.x*threads+tid]*corresp_list_length) + idx];
			tmp = input_pos[view_target*640*480+pidx];
			points[threads*3 + tid] = tmp.x;
			points[threads*4 + tid] = tmp.y;
			points[threads*5 + tid] = tmp.z;


			unsigned int wid = tid/WARP_SIZE;
			unsigned int wtid = tid - wid * WARP_SIZE;
			unsigned int gid = wtid/group_length;
			unsigned int gtid = wtid - gid * group_length;




			for(int d=0;d<6;d++)
			{

				buffer[threadIdx.x] = points[d*blockDim.x+threadIdx.x];
				__syncthreads();

				volatile float *warpLine = &buffer[wid*WARP_SIZE];

				if(gtid < 16){
					unsigned int posb = gid * group_length;
					if(groups_per_warp<=1) 	warpLine[posb + gtid] += warpLine[posb + gtid + 16];
					if(groups_per_warp<=2) 	warpLine[posb + gtid] += warpLine[posb + gtid + 8];
					if(groups_per_warp<=4) 	warpLine[posb + gtid] += warpLine[posb + gtid + 4];
					if(groups_per_warp<=8) 	warpLine[posb + gtid] += warpLine[posb + gtid + 2];
					if(groups_per_warp<=16) warpLine[posb + gtid] += warpLine[posb + gtid + 1];

					if(gtid<groups_per_warp)
					{
						centroids[d*n_matrices+wid*groups_per_warp+gtid] = warpLine[gtid*group_length]/group_length;
					}
				}
			}
			__syncthreads();

			points[threads*0 + threadIdx.x] -= centroids[0*n_matrices+wid*groups_per_warp+gid];
			points[threads*1 + threadIdx.x] -= centroids[1*n_matrices+wid*groups_per_warp+gid];
			points[threads*2 + threadIdx.x] -= centroids[2*n_matrices+wid*groups_per_warp+gid];

			points[threads*3 + threadIdx.x] -= centroids[3*n_matrices+wid*groups_per_warp+gid];
			points[threads*4 + threadIdx.x] -= centroids[4*n_matrices+wid*groups_per_warp+gid];
			points[threads*5 + threadIdx.x] -= centroids[5*n_matrices+wid*groups_per_warp+gid];


			__syncthreads();
			for(int sd=0;sd<3;sd++)
			{
				for(int td=0;td<3;td++)
				{

					buffer[threadIdx.x] = points[sd*threads+threadIdx.x] * points[(3+td)*threads+threadIdx.x];
					__syncthreads();

					volatile float *warpLine = &buffer[wid*WARP_SIZE];
					if(gtid < WARP_SIZE/2){
						unsigned int posb = gid * group_length;
						if(groups_per_warp<=1) 	warpLine[posb + gtid] += warpLine[posb + gtid + 16];
						if(groups_per_warp<=2) 	warpLine[posb + gtid] += warpLine[posb + gtid + 8];
						if(groups_per_warp<=4) 	warpLine[posb + gtid] += warpLine[posb + gtid + 4];
						if(groups_per_warp<=8) 	warpLine[posb + gtid] += warpLine[posb + gtid + 2];
						if(groups_per_warp<=16) warpLine[posb + gtid] += warpLine[posb + gtid + 1];


						if(gtid<groups_per_warp)
						{
							matricesH[(sd*3+td)*n_matrices+wid*groups_per_warp+gtid] = warpLine[gtid*group_length];
//							output_correlationMatrixes[ (sd*3+td)*n_rsac+blockIdx.x*n_matrices+wid*groups_per_warp+gtid] = warpLine[gtid*group_length];

						}
					}

				}


			}

			__syncthreads();
*/
		}

	};
	__global__ void estimateTransformations(const TransformationMatrixEstimator tfe){ tfe(); }
}


device::CorrespondenceListEstimator correspondanceList;
device::TransformationMatrixEstimator transformEstimator;

void
RigidBodyTransformationAdvancedEstimatior::init()
{

	correspondanceList.input_feature_histo = (float *)getInputDataPointer(HistogramMap);
	correspondanceList.input_idxList = (int *)getInputDataPointer(IdxList);
	correspondanceList.input_idxLength = (unsigned int *)getInputDataPointer(InfoList);

	transformEstimator.input_pos = (float4 *)getInputDataPointer(Coordiantes);
}

void
RigidBodyTransformationAdvancedEstimatior::execute()
{

	unsigned int view_combinations = ((n_view-1)*n_view)/2;

	printf("combinations: %d \n",view_combinations);

	thrust::device_vector<float> d_rndSList(s*view_combinations);
	correspondanceList.input_rndList = thrust::raw_pointer_cast(d_rndSList.data());

	thrust::device_vector<unsigned int> d_sIdxList(s*view_combinations);
	thrust::device_vector<float> d_corresp_prob(s*correspondanceList.corresp_list_length*view_combinations);
	thrust::device_vector<unsigned int> d_corresp_idx(s*correspondanceList.corresp_list_length*view_combinations);

	correspondanceList.output_s_idxList = thrust::raw_pointer_cast(d_sIdxList.data());
	correspondanceList.output_corresp_prob = thrust::raw_pointer_cast(d_corresp_prob.data());
	correspondanceList.output_corresp_idx = thrust::raw_pointer_cast(d_corresp_idx.data());


	correspondanceList.n_view = n_view;
	dim3 correspBlock(correspondanceList.dx);
	dim3 correspGrid(s/correspondanceList.local_points_per_block,1,view_combinations);

	hiprandGenerator_t gen ;
	checkCudaErrors(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
	checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL));

	checkCudaErrors(hiprandGenerateUniform(gen,correspondanceList.input_rndList,correspGrid.x*correspondanceList.local_points_per_block*view_combinations));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	device::estimateBestCorrespondences<<<correspGrid,correspBlock>>>(correspondanceList);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	thrust::host_vector<float> h_corresp_prob = d_corresp_prob;
	thrust::host_vector<unsigned int> h_cooresp_idx = d_corresp_idx;

	for(int i=0;i<8;i++)
	{
		printf("%d :",i);
		for(int k=0;k<correspondanceList.corresp_list_length;k++)
		{
			printf("%f | ",h_corresp_prob[k*s+i]);
		}
		printf("\n");
//		for(int k=0;k<correspondanceList.corresp_list_length;k++)
//		{
//			printf("%d | ",h_cooresp_idx[k*s+i]);
//		}
//		printf("\n");
	}


	transformEstimator.input_src_idx_corresp = correspondanceList.output_s_idxList;
	transformEstimator.input_target_idx_corresp = correspondanceList.output_corresp_idx;
	transformEstimator.input_target_prob_corresp = correspondanceList.output_corresp_prob;
	transformEstimator.n_view = n_view;
	transformEstimator.s_length = s;

	device::estimateTransformations<<<1,transformEstimator.threads>>>(transformEstimator);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



//	thrust::host_vector<float> h_rndSList = d_rndSList;
//
//	for(int i=0;i<s*view_combinations;i+=8)
//	{
//		printf("%d -> %f \n",i,h_rndSList.data()[i]);
//	}

}



RigidBodyTransformationAdvancedEstimatior::RigidBodyTransformationAdvancedEstimatior(unsigned int n_view, unsigned int rn, unsigned int s,unsigned int k) : n_view(n_view), rn(rn), s(s), k(k)
{
	DeviceDataParams transformationmatrixesParams;
	transformationmatrixesParams.elements = rn * ((n_view-1)*n_view)/2;
	transformationmatrixesParams.element_size = 12 * sizeof(float);
	transformationmatrixesParams.elementType = FLOAT1;
	transformationmatrixesParams.dataType = Matrix;
	addTargetData(addDeviceDataRequest(transformationmatrixesParams),TransformationMatrices);

	DeviceDataParams transformationMetaDataList;
	transformationMetaDataList.elements = ((n_view-1)*n_view)/2;
	transformationMetaDataList.element_size = sizeof(int);
	transformationMetaDataList.elementType = TransformationInfoListItem;
	transformationMetaDataList.dataType = Indice;
	addTargetData(addDeviceDataRequest(transformationMetaDataList),TransformationMetaDataList);
}

void RigidBodyTransformationAdvancedEstimatior::TestRBTAFct()
{
//	correspondanceList.bins_n_meta;

	n_view = 4;

	unsigned int s = 32;
	unsigned int view_combinations = ((n_view-1)*n_view)/2;

	printf("combinations: %d \n",view_combinations);

	hiprandGenerator_t gen ;
	checkCudaErrors(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
	checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL));

	thrust::device_vector<float> d_histos(n_view*640*480*correspondanceList.bins_n_meta);
	thrust::device_vector<int> d_idxList(n_view*640*480);
	thrust::device_vector<unsigned int> d_idxLength(n_view);
	thrust::device_vector<float> d_rndSList(s*view_combinations);

	correspondanceList.input_feature_histo = thrust::raw_pointer_cast(d_histos.data());
	correspondanceList.input_idxList = thrust::raw_pointer_cast(d_idxList.data());
	correspondanceList.input_idxLength = thrust::raw_pointer_cast(d_idxLength.data());
	correspondanceList.input_rndList = thrust::raw_pointer_cast(d_rndSList.data());

	correspondanceList.n_view = n_view;
	dim3 TestBlock(correspondanceList.dx);
	dim3 TestGrid(s/correspondanceList.local_points_per_block,1,view_combinations);


	checkCudaErrors(hiprandGenerateUniform(gen,correspondanceList.input_rndList,TestGrid.x*correspondanceList.local_points_per_block*view_combinations));
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


//	correspondanceList.

	device::estimateBestCorrespondences<<<TestGrid,TestBlock>>>(correspondanceList);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());



}

RigidBodyTransformationAdvancedEstimatior::~RigidBodyTransformationAdvancedEstimatior()
{
	// TODO Auto-generated destructor stub
}


