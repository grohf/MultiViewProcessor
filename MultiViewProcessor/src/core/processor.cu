

#include <vector>

// includes, cuda
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_cuda_gl.h>

#include "processor.h"


Processor::Processor()
{

//	findCudaGLDevice(0,(const char**)"");

}


void Processor::addFilter(Filter& filter)
{
	Enhancer *enh = &filter;
	EnhancerPtr ePtr(enh);
	enhancerPtrList.push_back(ePtr);
}

void Processor::start()
{
	findCudaDevice(0,(const char**)"");

	allocateDeviceMemory();

	/* INITS */
	srcPtr->init();
	for(int i=0;i<enhancerPtrList.size();i++)
	{
		enhancerPtrList[i]->init();
	}

	/* EXECUTES */
	srcPtr->execute();
	for(int i=0;i<enhancerPtrList.size();i++)
	{
		enhancerPtrList[i]->execute();
	}
}

