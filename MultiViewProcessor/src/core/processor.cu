

#include <vector>

// includes, cuda
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_cuda_gl.h>

#include "processor.h"


bool Processor::contRun;
bool Processor::contTry;
bool Processor::isAligned;

Processor::Processor()
{

//	findCudaGLDevice(0,(const char**)"");
	contRun = true;
	contTry = true;

	isAligned = false;
}


void Processor::addFilter(Filter& filter)
{
	Enhancer *enh = &filter;
	EnhancerPtr ePtr(enh);
	alignmentPtrList.push_back(ePtr);
}

void Processor::start()
{
	findCudaDevice(0,(const char**)"");

	allocateDeviceMemory();

	/* INITS */
	srcPtr->init();
	for(int i=0;i<alignmentPtrList.size();i++)
	{
		alignmentPtrList[i]->init();
	}

	for(int l=0;l<25 && contRun;l++)
//	while(contRun)
	{
		/* EXECUTES */
		srcPtr->execute();
		for(int i=0;i<alignmentPtrList.size()&&contTry;i++)
		{
			alignmentPtrList[i]->execute();
		}
	}
}

