/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <processor.h>

// includes, cuda
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_cuda_gl.h>

#include "../sources/SyncFreenectSource.h"
#include "../filter/TestFilter.h"
#include "../filter/TestFilter2.h"
#include "../filter/ATrousFilter.h"
#include "../feature/NormalPCAEstimator.h"
#include "../feature/FPFH.h"
#include "../feature/SVDEstimatorCPU.h"
#include "../feature/RigidBodyTransformationEstimator.h"


void runTestProcessor()
{
	SyncFreenectSource *src = new SyncFreenectSource();
//	SourcePtr src(new SyncFreenectSource());

	Processor p;
	p.setSource(SourcePtr(src));

//	SVDEstimator_CPU *svd_cpu = new SVDEstimator_CPU();
//	p.addFeature(svd_cpu);

	ATrousFilter *atrousfilter = new ATrousFilter();
	atrousfilter->setInput2DPointCloud(src->getTargetData(SyncFreenectSource::PointXYZI));
	atrousfilter->setInputSensorInfo(src->getTargetData(SyncFreenectSource::SensorInfoList));

	p.addFilter(FilterPtr(atrousfilter));

	NormalPCAEstimator *nPCAestimator = new NormalPCAEstimator();
	nPCAestimator->setWorldCoordinates(atrousfilter->getFilteredWorldCoordinates());
//	nPCAestimator->setWorldCoordinates(src->getTargetData(SyncFreenectSource::PointXYZI));
	p.addFeature(nPCAestimator);

	FPFH *fpfhEstimator = new FPFH();
	fpfhEstimator->setPointCoordinates(atrousfilter->getFilteredWorldCoordinates());
	fpfhEstimator->setNormals(nPCAestimator->getNormals());

	p.addFeature(fpfhEstimator);

	RigidBodyTransformationEstimator *rbEstimator = new RigidBodyTransformationEstimator(1024,32);
	rbEstimator->setPersistanceHistogramMap(fpfhEstimator->getFPFH());
	rbEstimator->setPersistanceIndexList(fpfhEstimator->getPersistanceIndexList());
	rbEstimator->setPersistenceInfoList(fpfhEstimator->getPersistenceInfoList());
	p.addFeature(rbEstimator);

//	FilterPtr fp = atrousfilter->ptr;

	p.start();

	src->~SyncFreenectSource();

}

void coorespTest()
{
	SyncFreenectSource *src = new SyncFreenectSource();
//	SourcePtr src(new SyncFreenectSource());

	Processor p;
	p.setSource(SourcePtr(src));
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char **argv) {

	runTestProcessor();
	return 0;
}
