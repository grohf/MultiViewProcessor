/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <processor.h>

// includes, cuda
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_cuda_gl.h>

#include "../sources/SyncFreenectSource.h"
#include "../filter/TestFilter.h"

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char **argv) {

//	TestFilter *f = new TestFilter();

//	findCudaGLDevice(argc,(const char**)argv);

//	findCudaDevice(argc,(const char**)argv);

	SyncFreenectSource src;

	Processor p;
	p.addSource(src);

	return 0;
}
